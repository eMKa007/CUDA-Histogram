#include "hip/hip_runtime.h"
#include "../Public/HistGPU.h"

HistGPU::HistGPU(int* inputArray_in, int inputArraySize_in, int* HistogramGPU_in) :
	inputArray(inputArray_in), inputArraySize(inputArraySize_in), HistogramGPU(HistogramGPU_in)
{
	if( !inputArraySize_in || 0 == inputArraySize_in || !HistogramGPU_in )
		throw std::invalid_argument("HistCPU class: Received invalid argument in constructor.");
}
HistGPU::~HistGPU()
{

}

float HistGPU::RunSingleTest_GPU()
{
	int* dev_inputArray = nullptr;
	int* dev_Histogram = nullptr;
	hipError_t cudaStatus;

	hipEventRecord(start);

	//Allocate space on GPU.
	cudaStatus = hipMalloc((void**)&dev_inputArray, inputArraySize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc() fail! Can not allocate memory on GPU.\n");
		exit(-1);
	}

	cudaStatus = hipMalloc((void**)&dev_Histogram, 256 * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc() fail! Can not allocate memory on GPU.\n");
		exit(-1);
	}

	// Initialize device Histogram with 0
	cudaStatus = hipMemset(dev_Histogram, 0, 256 * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemset() fail! Can not set memory on GPU.\n");
		exit(-1);
	}

	// Copy input to previously allocated memory on GPU.
	cudaStatus = hipMemcpy(dev_inputArray, inputArray, inputArraySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy() fail! Can not copy data to GPU device.\n");
		exit(-1);
	}

	//Check available number of multiprocessors on GPU device- it will be used in kernel function.
	hipDeviceProp_t properties;
	cudaStatus = hipGetDeviceProperties(&properties, 0);
	if (cudaStatus != hipSuccess)
	{
		printf("hipGetDeviceProperties() fail.");
		exit(-1);
	}

	//Launch kernel. ==============================================================================
	int blocks = properties.multiProcessorCount;
	GPU_Histogram_Kernel << <blocks * 2, 256 >> > (dev_inputArray, inputArraySize, dev_Histogram);

	// Check for kernel errors.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("GPU_Histogram() kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// Wait for kernel to finish work, and check for any errors during kernel work.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize() returned error code %d after launching!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(HistogramGPU, dev_Histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy() device to host failed!");
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float miliseconds = 0;
	hipEventElapsedTime(&miliseconds, start, stop);

	hipFree(dev_inputArray);
	hipFree(dev_Histogram);
	
	return miliseconds;
}



float HistGPU::Test_GPU(unsigned int NumberOfExec)
{
	hipError_t cudaStatus;

	//Assume, we will use first GPU device.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		printf("hipSetDevice() fail! Do you have CUDA available device?\n");
		exit(-1);
	}

	// Cuda events used to measure execution time.
	CreateTimeEvents();
	
	float TotalTime = 0;
	for (int TryNumber = 0; TryNumber < NumberOfExec; TryNumber++)
	{
		TotalTime += RunSingleTest_GPU();
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	return TotalTime/NumberOfExec;
}

void HistGPU::CreateTimeEvents()
{
	hipError_t cudaStatus;
	cudaStatus = hipEventCreate(&start);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create start event to measure execution time.\n");
		exit(-1);
	}

	cudaStatus = hipEventCreate(&stop);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create start event to measure execution time.\n");
		exit(-1);
	}
}

__global__ void GPU_Histogram_Kernel(int* inputArray, int inputArraySize, int* HistogramGPU)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while (i < inputArraySize)
	{
		atomicAdd(&HistogramGPU[inputArray[i]], 1);
		i += stride;
	}
}