#include "hip/hip_runtime.h"
#include "../Public/HistGPU.h"

HistGPU::HistGPU(int* inputArray_in, int inputArraySize_in, int* HistogramGPU_in) :
	inputArray(inputArray_in), inputArraySize(inputArraySize_in), HistogramGPU(HistogramGPU_in)
{
	if( !inputArraySize_in || 0 == inputArraySize_in || !HistogramGPU_in )
		throw std::invalid_argument("HistCPU class: Received invalid argument in constructor.");
}
HistGPU::~HistGPU()
{

}

void HistGPU::RunSingleTest_GPU()
{
	int* dev_inputArray = nullptr;
	int* dev_Histogram = nullptr;
	hipError_t cudaStatus;

	hipEventRecord(beforeAlloc);

	//Allocate space on GPU.
	cudaStatus = hipMalloc((void**)&dev_inputArray, inputArraySize * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc() fail! Can not allocate memory on GPU.\n");
		throw(cudaStatus);
	}

	cudaStatus = hipMalloc((void**)&dev_Histogram, 256 * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMalloc() fail! Can not allocate memory on GPU.\n");
		throw(cudaStatus);
	}

	// Initialize device Histogram with 0
	cudaStatus = hipMemset(dev_Histogram, 0, 256 * sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		printf("hipMemset() fail! Can not set memory on GPU.\n");
		throw(cudaStatus);
	}

	// Copy input to previously allocated memory on GPU.
	cudaStatus = hipMemcpy(dev_inputArray, inputArray, inputArraySize * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy() fail! Can not copy data to GPU device.\n");
		throw(cudaStatus);
	}

	//Check available number of multiprocessors on GPU device- it will be used in kernel function.
	hipDeviceProp_t properties;
	cudaStatus = hipGetDeviceProperties(&properties, 0);
	if (cudaStatus != hipSuccess)
	{
		printf("hipGetDeviceProperties() fail.");
		throw(cudaStatus);
	}

	hipEventRecord(beforeCompute);

	//Launch kernel. ==============================================================================
	int blocks = properties.multiProcessorCount;
	GPU_Histogram_Kernel << <blocks * 2, 256 >> > (dev_inputArray, inputArraySize, dev_Histogram);

	hipEventRecord(afterCompute);
	hipEventSynchronize(afterCompute);

	// Check for kernel errors.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf("GPU_Histogram() kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// Wait for kernel to finish work, and check for any errors during kernel work.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		printf("hipDeviceSynchronize() returned error code %d after launching!\n", cudaStatus);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(HistogramGPU, dev_Histogram, 256 * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy() device to host failed!");
	}

	hipEventRecord(afterAlloc);
	hipEventSynchronize(afterAlloc);

	float withAllocation = 0;
	float woAllocation = 0;
	hipEventElapsedTime(&withAllocation, beforeAlloc, afterAlloc);
	hipEventElapsedTime(&woAllocation, beforeCompute, afterCompute);
	totalMiliseconds_withAllocation += withAllocation;
	totalMiliseconds_woAllocation += woAllocation;

	hipFree(dev_inputArray);
	hipFree(dev_Histogram);
}



void HistGPU::Test_GPU(unsigned int NumberOfExec)
{
	hipError_t cudaStatus;

	//Assume, we will use first GPU device.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		printf("hipSetDevice() fail! Do you have CUDA available device?\n");
		throw(cudaStatus);
	}

	// Cuda events used to measure execution time.
	CreateTimeEvents();
	
	for (int TryNumber = 0; TryNumber < NumberOfExec; TryNumber++)
	{
		RunSingleTest_GPU();
	}

	hipEventDestroy(beforeAlloc);
	hipEventDestroy(afterAlloc);
	hipEventDestroy(beforeCompute);
	hipEventDestroy(afterCompute);
	
	ComputeMeanTimes(NumberOfExec);
}

void HistGPU::CreateTimeEvents()
{
	hipError_t cudaStatus;
	hipEvent_t* Event[4] = { &beforeAlloc, &beforeCompute, &afterAlloc, &afterCompute };

	for (int i = 0; i < 4; i++)
	{
		cudaStatus = hipEventCreate(Event[i]);
		if (cudaStatus != hipSuccess) {
			printf("hipEventCreate() fail! Can not create beforeAlloc event to measure execution time.\n");
			throw(cudaStatus);
		}
	}

	/*cudaStatus = hipEventCreate(&beforeAlloc);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create beforeAlloc event to measure execution time.\n");
		throw(cudaStatus);
	}

	cudaStatus = hipEventCreate(&beforeCompute);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create beforeCompute event to measure execution time.\n");
		throw(cudaStatus);
	}

	cudaStatus = hipEventCreate(&afterAlloc);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create afterAlloc event to measure execution time.\n");
		throw(cudaStatus);
	}

	cudaStatus = hipEventCreate(&afterCompute);
	if (cudaStatus != hipSuccess) {
		printf("hipEventCreate() fail! Can not create afterCompute event to measure execution time.\n");
		throw(cudaStatus);
	}*/
}

void HistGPU::ComputeMeanTimes(unsigned int NumberOfExec)
{
	msWithAlloc = totalMiliseconds_withAllocation / NumberOfExec;
	msWithoutAlloc = totalMiliseconds_woAllocation / NumberOfExec;
}

__global__ void GPU_Histogram_Kernel(int* inputArray, int inputArraySize, int* HistogramGPU)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	while (i < inputArraySize)
	{
		atomicAdd(&HistogramGPU[inputArray[i]], 1);
		i += stride;
	}
}